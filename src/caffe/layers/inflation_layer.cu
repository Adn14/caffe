#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void inflate_gpu_kernel(const int n, const Dtype* data_bottom,
    const int height, const int width, const int inflate_size_h, const int inflate_size_w,
    const Dtype placeholder, const Dtype scale,
    const int inflate_rate_h, const int inflate_rate_w,
    Dtype* data_top) {
  CUDA_KERNEL_LOOP(index, n) {
	  int w_out = index % inflate_size_w;
	  int h_index = index / inflate_size_w;
	  int h_out = h_index % inflate_size_h;
	  int channel_out = h_index / inflate_size_h;

	  data_top[index] = ((w_out % inflate_rate_w == 0)&&(h_out % inflate_rate_h == 0))
			  	  	  	  ?data_bottom[(channel_out * height + h_out/inflate_rate_h)*width + w_out/inflate_rate_w] * scale
			  	  	  	  :placeholder;

  }
}

template <typename Dtype>
__global__ void shrink_gpu_kernel(const int n, const Dtype* data_top,
	    const int height, const int width, const int inflate_size_h, const int inflate_size_w,
	    const Dtype placeholder, const Dtype scale,
	    const int inflate_rate_h, const int inflate_rate_w,
	    Dtype* data_bottom) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_bt = index % width;
    int h_index = index/width;
    int h_bt = h_index % height;
    int channel_bt = h_index/height;

    data_bottom[index] = data_top[(channel_bt*inflate_size_h + h_bt*inflate_rate_h)*inflate_size_w + w_bt*inflate_rate_w] * scale;
  }
}

template <typename Dtype>
void InflationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  Forward_cpu(bottom, top);
	int num_kernels = channels_ * inflate_size_h_ * inflate_size_w_;
	const Dtype* data_bottom = bottom[0]->gpu_data();
	Dtype* data_top = (*top)[0]->mutable_gpu_data();

	for (int n = 0; n < bottom[0]->num(); n++){
		inflate_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
	                             CAFFE_CUDA_NUM_THREADS>>>(
	      num_kernels, data_bottom + bottom[0]->offset(n),
	      height_, width_, inflate_size_h_, inflate_size_w_,
	      placeholder_, scale_,
	      inflate_rate_h_, inflate_rate_w_,
	      data_top + (*top)[0]->offset(n)
	    );
		CUDA_POST_KERNEL_CHECK;
	}
}

template <typename Dtype>
void InflationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
//  Backward_cpu(top, propagate_down, bottom);
	int num_kernels = channels_ * height_ * width_; //launch as many threads as the pixel number in the bottom data
	Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
	const Dtype* top_diff = top[0]->gpu_diff();

	caffe_gpu_set<Dtype>((*bottom)[0]->count(), Dtype(0.), bottom_diff);

	for (int n = 0; n < (*bottom)[0]->num(); n++){
		shrink_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
								 CAFFE_CUDA_NUM_THREADS>>>(
		  num_kernels, top_diff + top[0]->offset(n),
		  height_, width_, inflate_size_h_, inflate_size_w_,
		  placeholder_, scale_,
		  inflate_rate_h_, inflate_rate_w_,
		  bottom_diff + (*bottom)[0]->offset(n)
		);
		CUDA_POST_KERNEL_CHECK;
	}

}


INSTANTIATE_CLASS(InflationLayer);

}  // namespace caffe
